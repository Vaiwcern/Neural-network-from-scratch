#include "hip/hip_runtime.h"
#include "DenseLayer.h"
#include "Kernel.h"
#include <random>
#include <cmath>
#include <cstring>
#include "Macro.h"

DenseLayer::DenseLayer(int input_size, int output_size, ActivationFunction *activation, int max_batch)
    : input_size(input_size), output_size(output_size), activation(activation), max_batch(max_batch)
{
    weights = new float[input_size * output_size];
    biases = new float[output_size];

    weight_gradients = new float[input_size * output_size];
    bias_gradients = new float[output_size];

    last_input = new float[input_size * max_batch];
    last_output = new float[output_size * max_batch];

    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> dist(0.0f, sqrtf(2.0f / input_size));

    for (int k = 0; k < input_size; ++k) {
        for (int i = 0; i < output_size; ++i)
        {
            for (int j = 0; j < input_size; ++j)
            {
                weights[i * input_size + j] = dist(gen);
            }
            biases[i] = 0.0f;
        }
    }

    // Allocate GPU memory once
    CHECK(hipMalloc(&d_input, input_size * max_batch * sizeof(float)));
    CHECK(hipMalloc(&d_output, output_size * max_batch * sizeof(float)));
    CHECK(hipMalloc(&d_weights, input_size * output_size * sizeof(float)));
    CHECK(hipMalloc(&d_biases, output_size * sizeof(float)));
    CHECK(hipMalloc(&d_linear_output, output_size * max_batch * sizeof(float)));
    CHECK(hipMalloc(&d_wgrad, input_size * output_size * sizeof(float)));
    CHECK(hipMalloc(&d_bgrad, output_size * sizeof(float)));
    CHECK(hipMalloc(&d_igrad, input_size * max_batch * sizeof(float)));
    CHECK(hipMalloc(&d_act, output_size * max_batch * sizeof(float)));

    // Copy initial weights, biases to device
    CHECK(hipMemcpy(d_weights, weights, input_size * output_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_biases, biases, output_size * sizeof(float), hipMemcpyHostToDevice));
}

// Forward batch
void DenseLayer::forward(float *input, float *output, int batch_size)
{
    memcpy(last_input, input, input_size * batch_size * sizeof(float));

    // Copy input to device once
    CHECK(hipMemcpy(d_input, input, input_size * batch_size * sizeof(float), hipMemcpyHostToDevice));

    int threads = 512; // đảm bảo threads >= output_size
    int blocks = batch_size;
    size_t shared_mem_size = threads * sizeof(float); // vì ta dùng s_input[threads]
    forward_kernel<<<blocks, threads, shared_mem_size>>>(d_input, d_linear_output, d_weights, d_biases,
                                                         input_size, output_size, batch_size);
    CHECK(hipDeviceSynchronize());

    // linear_output đã ở d_linear_output, apply activation trên host hoặc device
    // Ở đây ta làm activation trên host->device->host không hiệu quả,
    // ta nên implement activation kernel và gọi trực tiếp trên device.
    // Giả sử activation->activate(...) đã sử dụng kernel như ReLU hay softmax:
    // Ta gọi thẳng activation kernel trên d_linear_output -> d_output

    if (dynamic_cast<ReLU *>(activation))
    {
        int size = output_size * batch_size;
        int threads_act = 256;
        int blocks_act = (size + threads_act - 1) / threads_act;
        relu_kernel<<<blocks_act, threads_act>>>(d_linear_output, d_output, size);
        CHECK(hipDeviceSynchronize());
    }
    else if (dynamic_cast<Softmax *>(activation))
    {
        // Softmax trên batch_size mẫu: Mỗi mẫu là 1 vector.
        for (int b = 0; b < batch_size; b++)
        {
            softmax_kernel<<<1, output_size, 2 * output_size * sizeof(float)>>>(d_linear_output + b * output_size, d_output + b * output_size, output_size);
        }
        CHECK(hipDeviceSynchronize());
    }

    // Copy output về host
    CHECK(hipMemcpy(output, d_output, output_size * batch_size * sizeof(float), hipMemcpyDeviceToHost));
    memcpy(last_output, output, output_size * batch_size * sizeof(float));
}

// Backward batch
void DenseLayer::backward(float *output_gradient, float *input_gradient, int batch_size)
{
    // d_act = output_gradient trên device
    CHECK(hipMemcpy(d_act, output_gradient, output_size * batch_size * sizeof(float), hipMemcpyHostToDevice));

    // Gọi derivative kernel:
    int size = output_size * batch_size;
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    // Chúng ta đã có last_output trên host, cần copy lên device để derivative.
    CHECK(hipMemcpy(d_output, last_output, output_size * batch_size * sizeof(float), hipMemcpyHostToDevice));
    relu_derivative_kernel<<<blocks, threads>>>(d_output, d_act, size);
    CHECK(hipDeviceSynchronize());

    // bây giờ d_act là output_gradient sau activation derivative
    CHECK(hipMemcpy(d_input, last_input, input_size * batch_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemset(d_wgrad, 0, input_size * output_size * sizeof(float)));
    CHECK(hipMemset(d_bgrad, 0, output_size * sizeof(float)));
    CHECK(hipMemset(d_igrad, 0, input_size * batch_size * sizeof(float)));

    int backward_blocks = output_size;
    int backward_threads = input_size;
    size_t shared_mem_size = input_size * sizeof(float); // Shared memory for weights

    backward_kernel<<<backward_blocks, backward_threads, shared_mem_size>>>(
        d_input,
        d_act,
        d_weights,
        d_wgrad,
        d_bgrad,
        d_igrad,
        input_size,
        output_size,
        batch_size);
    CHECK(hipDeviceSynchronize());

    // Copy gradients về host
    CHECK(hipMemcpy(weight_gradients, d_wgrad, input_size * output_size * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(bias_gradients, d_bgrad, output_size * sizeof(float), hipMemcpyDeviceToHost));

    if (input_gradient)
    {
        CHECK(hipMemcpy(input_gradient, d_igrad, input_size * batch_size * sizeof(float), hipMemcpyDeviceToHost));
    }
}

// Update weights
void DenseLayer::update_weights(float learning_rate, int batch_size)
{
    // Copy host gradients lên GPU rồi update tại chỗ
    CHECK(hipMemcpy(d_wgrad, weight_gradients, input_size * output_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bgrad, bias_gradients, output_size * sizeof(float), hipMemcpyHostToDevice));

    int threads = 512;
    int blocks = (output_size + threads - 1) / threads;
    float lr = learning_rate / (float)batch_size;
    update_weights_kernel<<<blocks, threads>>>(d_weights, d_wgrad, d_biases, d_bgrad, lr, input_size, output_size);
    CHECK(hipDeviceSynchronize());

    // Copy weights, biases về host (nếu cần)
    CHECK(hipMemcpy(weights, d_weights, input_size * output_size * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(biases, d_biases, output_size * sizeof(float), hipMemcpyDeviceToHost));
}

DenseLayer::~DenseLayer()
{
    delete[] weights;
    delete[] biases;
    delete[] weight_gradients;
    delete[] bias_gradients;
    delete[] last_input;
    delete[] last_output;
    delete activation;

    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
    CHECK(hipFree(d_weights));
    CHECK(hipFree(d_biases));
    CHECK(hipFree(d_linear_output));
    CHECK(hipFree(d_wgrad));
    CHECK(hipFree(d_bgrad));
    CHECK(hipFree(d_igrad));
    CHECK(hipFree(d_act));
}
