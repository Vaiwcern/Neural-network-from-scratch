#include "DenseLayer.h"
#include "CudaHelper.h"
#include <cstdlib>
#include <ctime>
#include <iostream>

using namespace std;

// Constructor
DenseLayer::DenseLayer(int input_size, int output_size, ActivationFunction* activation)
    : input_size(input_size), output_size(output_size), activation(activation) 
{
    // Cấp phát bộ nhớ cho weights và biases
    weights = new float[input_size * output_size];
    biases = new float[output_size];

    // Khởi tạo ngẫu nhiên trọng số và độ chệch
    srand(time(0));
    for (int i = 0; i < output_size; ++i) {
        for (int j = 0; j < input_size; ++j) {
            weights[i * input_size + j] = ((float)rand() / RAND_MAX) * 2 - 1;  // Random between -1 and 1
        }
        biases[i] = ((float)rand() / RAND_MAX) * 2 - 1;
    }
}

// Phương thức forward pass
void DenseLayer::forward(float* input, float* output) {
    cout << "FORWARD LẦN 1:" << endl;
    cout << "Input size: " << input_size << ", Output size: " << output_size << endl;
    cout << "Weights matrix (input_size x output_size): " << input_size << " x " << output_size << endl;

    // Print input values for debugging
    cout << "Input: ";
    for (int i = 0; i < input_size; ++i) {
        cout << input[i] << " ";
    }
    cout << endl;

    // Print weights matrix (for debugging)
    cout << "Weights (Matrix):" << endl;
    for (int i = 0; i < output_size; ++i) {  // Loop through rows (output_size)
        for (int j = 0; j < input_size; ++j) {  // Loop through columns (input_size)
            cout << weights[i * input_size + j] << " ";  // Print element at [i, j]
        }
        cout << endl;  // New line after each row
    }

    // Memory allocation and CUDA code for forward pass
    float *d_input, *d_output, *d_weights, *d_biases;

    CHECK(hipMalloc(&d_input, input_size * sizeof(float)));
    CHECK(hipMalloc(&d_output, output_size * sizeof(float)));
    CHECK(hipMalloc(&d_weights, input_size * output_size * sizeof(float)));
    CHECK(hipMalloc(&d_biases, output_size * sizeof(float)));

    // Copy data to device
    CHECK(hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_weights, weights, input_size * output_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_biases, biases, output_size * sizeof(float), hipMemcpyHostToDevice));

    // Perform the forward pass with kernel
    int blocks = (output_size + 255) / 256;
    forward_kernel<<<blocks, 256>>>(d_input, d_output, d_weights, d_biases, input_size, output_size);
    CHECK(hipGetLastError());  // Check for kernel launch errors
    CHECK(hipDeviceSynchronize());  // Ensure kernel execution is finished

    cout << "-------HIHI---------" << "\n";

    // Apply activation function (ReLU or Softmax)
    activation->activate(d_output, d_output, output_size);

    cout << "-------HIHI---------" << "\n";

    // Copy result back to host
    CHECK(hipMemcpy(output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost));

    // Print output for debugging
    cout << "Output: ";
    for (int i = 0; i < output_size; ++i) {
        cout << output[i] << " ";
    }
    cout << endl;

    // Free memory
    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
    CHECK(hipFree(d_weights));
    CHECK(hipFree(d_biases));
}


// Phương thức backward pass (tính toán gradient)
void DenseLayer::backward(float* input, float* output_gradient, float* weight_gradients, float* bias_gradients, int batch_size) {
    float *d_input, *d_output_gradient, *d_weights, *d_weight_gradients, *d_bias_gradients;

    // Cấp phát bộ nhớ trên device (GPU)
    CHECK(hipMalloc(&d_input, input_size * sizeof(float)));
    CHECK(hipMalloc(&d_output_gradient, output_size * sizeof(float)));
    CHECK(hipMalloc(&d_weights, input_size * output_size * sizeof(float)));
    CHECK(hipMalloc(&d_weight_gradients, input_size * output_size * sizeof(float)));
    CHECK(hipMalloc(&d_bias_gradients, output_size * sizeof(float)));

    // Sao chép dữ liệu từ host (CPU) vào device (GPU)
    CHECK(hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_output_gradient, output_gradient, output_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_weights, weights, input_size * output_size * sizeof(float), hipMemcpyHostToDevice));

    // Gọi kernel backward để tính gradient đối với trọng số và độ chệch
    int blocks = (output_size + 255) / 256;
    backward_kernel<<<blocks, 256>>>(d_input, d_output_gradient, d_weights, d_weight_gradients, d_bias_gradients, input_size, output_size);
    CHECK(hipDeviceSynchronize());

    // Sao chép gradient về host
    CHECK(hipMemcpy(weight_gradients, d_weight_gradients, input_size * output_size * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(bias_gradients, d_bias_gradients, output_size * sizeof(float), hipMemcpyDeviceToHost));

    // Giải phóng bộ nhớ trên device
    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output_gradient));
    CHECK(hipFree(d_weights));
    CHECK(hipFree(d_weight_gradients));
    CHECK(hipFree(d_bias_gradients));
}

// Phương thức cập nhật trọng số
void DenseLayer::update_weights(float* weight_gradients, float* bias_gradients, float learning_rate, int batch_size) {
    float *d_weights, *d_weight_gradients, *d_biases, *d_bias_gradients;

    // Cấp phát bộ nhớ trên device (GPU)
    CHECK(hipMalloc(&d_weights, input_size * output_size * sizeof(float)));
    CHECK(hipMalloc(&d_weight_gradients, input_size * output_size * sizeof(float)));
    CHECK(hipMalloc(&d_biases, output_size * sizeof(float)));
    CHECK(hipMalloc(&d_bias_gradients, output_size * sizeof(float)));

    // Sao chép dữ liệu từ host (CPU) vào device (GPU)
    CHECK(hipMemcpy(d_weights, weights, input_size * output_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_weight_gradients, weight_gradients, input_size * output_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_biases, biases, output_size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_bias_gradients, bias_gradients, output_size * sizeof(float), hipMemcpyHostToDevice));

    // Gọi kernel update_weights để cập nhật trọng số và độ chệch
    int blocks = (output_size + 255) / 256;
    update_weights_kernel<<<blocks, 256>>>(d_weights, d_weight_gradients, d_biases, d_bias_gradients, learning_rate, input_size, output_size);
    CHECK(hipDeviceSynchronize());

    // Sao chép trọng số và độ chệch đã cập nhật về host
    CHECK(hipMemcpy(weights, d_weights, input_size * output_size * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(biases, d_biases, output_size * sizeof(float), hipMemcpyDeviceToHost));

    // Giải phóng bộ nhớ trên device
    CHECK(hipFree(d_weights));
    CHECK(hipFree(d_weight_gradients));
    CHECK(hipFree(d_biases));
    CHECK(hipFree(d_bias_gradients));
}

// Destructor
DenseLayer::~DenseLayer() {
    delete[] weights;
    delete[] biases;
}
