
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>  // Đảm bảo bạn bao gồm thư viện vector

using namespace std;  

// Hàm để chuẩn hóa dữ liệu hình ảnh từ unsigned char sang float
void normalize_data(vector<unsigned char>& images, float* output, int num_samples, int image_size) {
    for (int i = 0; i < num_samples; ++i) {
        for (int j = 0; j < image_size; ++j) {
            // Chuyển đổi từ unsigned char (0-255) sang float (0.0 - 1.0)
            output[i * image_size + j] = images[i * image_size + j] / 255.0f;
        }
    }
}

void normalize_labels(vector<unsigned char>& labels, float* output, int num_samples) {
    for (int i = 0; i < num_samples; ++i) {
        int label = labels[i];
        for (int j = 0; j < 10; ++j) {
            output[i * 10 + j] = (j == label) ? 1.0f : 0.0f;
        }
    }
}
