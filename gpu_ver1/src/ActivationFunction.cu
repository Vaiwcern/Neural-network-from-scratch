#include "ActivationFunction.h"
#include "Kernel.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include "Macro.h"

void ReLU::activate(float* input, float* output, int size) const {
    float *d_input, *d_output;

    CHECK(hipMalloc(&d_input, size * sizeof(float)));
    CHECK(hipMalloc(&d_output, size * sizeof(float)));

    CHECK(hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice));

    int blocks = (size + 255) / 256;
    relu_kernel<<<blocks, 256>>>(d_input, d_output, size);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
}

void ReLU::derivative(float* output, float* d_output, int size) const {
    float *d_out, *d_dout;

    CHECK(hipMalloc(&d_out, size * sizeof(float)));
    CHECK(hipMalloc(&d_dout, size * sizeof(float)));

    CHECK(hipMemcpy(d_out, output, size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_dout, d_output, size * sizeof(float), hipMemcpyHostToDevice));

    int blocks = (size + 255) / 256;
    relu_derivative_kernel<<<blocks, 256>>>(d_out, d_dout, size);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(d_output, d_dout, size * sizeof(float), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_out));
    CHECK(hipFree(d_dout));
}

void Softmax::activate(float* input, float* output, int size) const {
    float *d_input, *d_output;

    CHECK(hipMalloc(&d_input, size * sizeof(float)));
    CHECK(hipMalloc(&d_output, size * sizeof(float)));

    CHECK(hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice));

    softmax_kernel<<<1, 1>>>(d_input, d_output, size);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
}

void Softmax::derivative(float* output, float* d_output, int size) const {
    // Softmax + CrossEntropy: derivative trực tiếp = output - target (đã xử lý bên ANN::backward)
    // Không làm gì ở đây
}