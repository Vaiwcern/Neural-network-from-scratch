#include "ActivationFunction.h"
#include "Kernel.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>

// Phương thức kích hoạt ReLU cho toàn bộ vector
void ReLU::activate(float* input, float* output, int size) const {
    float *d_input, *d_output;

    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    int blocks = (size + 255) / 256;
    relu_kernel<<<blocks, 256>>>(d_input, d_output, size);
    
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void ReLU::derivative(float* output, float* d_output, int size) const {
    // output ở đây là output sau ReLU
    float *d_out, *d_dout;
    hipMalloc(&d_out, size * sizeof(float));
    hipMalloc(&d_dout, size * sizeof(float));

    hipMemcpy(d_out, output, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dout, d_output, size * sizeof(float), hipMemcpyHostToDevice);

    int blocks = (size + 255) / 256;
    relu_derivative_kernel<<<blocks,256>>>(d_out, d_dout, size);
    hipDeviceSynchronize();

    hipMemcpy(d_output, d_dout, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_out);
    hipFree(d_dout);
}

// Phương thức kích hoạt Softmax cho toàn bộ vector
void Softmax::activate(float* input, float* output, int size) const {
    float *d_input, *d_output;

    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    softmax_kernel<<<1, 1>>>(d_input, d_output, size);
    hipDeviceSynchronize();

    hipDeviceSynchronize();

    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

void Softmax::derivative(float* output, float* d_output, int size) const {
    // Với Softmax + CrossEntropy, derivative thường được tính trực tiếp: d_output = output - target
    // Ở đây ta không cần kernel riêng.
    // Chỉ để tuân thủ interface
    // Không làm gì ở đây.
}