#include "ActivateFunction.h"
#include <cmath>
#include <hip/hip_runtime.h>

// Phương thức kích hoạt ReLU cho toàn bộ vector
void ReLU::activate(float* input, float* output, int size) const {
    float *d_input, *d_output;

    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    int blocks = (size + 255) / 256;
    relu_kernel<<<blocks, 256>>>(d_input, d_output, size);
    
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

// Phương thức kích hoạt Softmax cho toàn bộ vector
void Softmax::activate(float* input, float* output, int size) const {
    float *d_input, *d_output;

    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    int blocks = (size + 255) / 256;
    softmax_kernel<<<blocks, 256>>>(d_input, d_output, size);

    hipDeviceSynchronize();

    hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
