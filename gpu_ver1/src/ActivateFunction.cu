#include "ActivateFunction.h"
#include <cmath>
#include <iostream>
#include <vector>
#include <limits>
#include <hip/hip_runtime.h>
#include "CudaHelper.h"

// Phương thức kích hoạt ReLU cho toàn bộ vector
void ReLU::activate(float* input, float* output, int size) const {
    float *d_input, *d_output;

    CHECK(hipMalloc(&d_input, size * sizeof(float)));
    CHECK(hipMalloc(&d_output, size * sizeof(float)));

    CHECK(hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice));

    int blocks = (size + 255) / 256;
    relu_kernel<<<blocks, 256>>>(d_input, d_output, size);
    
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
}

void Softmax::activate(float* input, float* output, int size) const {
    // Tạo bộ nhớ trên device
    float* d_input;
    float* d_output;

    CHECK(hipMalloc(&d_input, size * sizeof(float)));
    CHECK(hipMalloc(&d_output, size * sizeof(float)));

    // Sao chép dữ liệu từ host vào device
    CHECK(hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice));

    // Thực thi kernel
    int block_size = 256;  // Kích thước block
    int num_blocks = (size + block_size - 1) / block_size;  // Tính số block

    softmax_kernel<<<num_blocks, block_size>>>(d_input, d_output, size);
    CHECK(hipDeviceSynchronize());  // Đồng bộ hóa để đảm bảo kernel đã hoàn thành

    // Sao chép kết quả từ device về host
    CHECK(hipMemcpy(output, d_output, size * sizeof(float), hipMemcpyDeviceToHost));

    // Giải phóng bộ nhớ trên device
    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
}

