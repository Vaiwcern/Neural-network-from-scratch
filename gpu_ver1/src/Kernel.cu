#include "hip/hip_runtime.h"
#include "Kernel.h"
#include <cmath>

__global__ void forward_kernel(float *input, float *output, float *weights, float *biases, int input_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < output_size) {
        // Tính toán tổng có trọng số (linear transformation)
        float sum = 0.0f;
        for (int j = 0; j < input_size; ++j) {
            sum += weights[idx * input_size + j] * input[j];
        }
        sum += biases[idx];
        output[idx] = sum;  // Không áp dụng hàm kích hoạt ở đây
    }
}

__global__ void relu_kernel(float *input, float *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Áp dụng hàm kích hoạt ReLU
        output[idx] = (input[idx] > 0) ? input[idx] : 0;  // ReLU: f(x) = max(0, x)
    }
}

__global__ void softmax_kernel(float *input, float *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Tính toán Softmax cho toàn bộ output
        float max_val = -INFINITY;
        for (int i = 0; i < size; ++i) {
            max_val = max(max_val, input[i]);
        }

        // Tính e^(input[i] - max_val)
        float sum_exp = 0.0f;
        for (int i = 0; i < size; ++i) {
            output[i] = exp(input[i] - max_val);
            sum_exp += output[i];
        }

        // Chuẩn hóa Softmax
        output[idx] /= sum_exp;
    }
}

__global__ void backward_kernel(float *input, float *output_gradient, float *weights, float *weight_gradients, float *bias_gradients, int input_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < output_size) {
        for (int j = 0; j < input_size; ++j) {
            weight_gradients[idx * input_size + j] = output_gradient[idx] * input[j];
        }
        bias_gradients[idx] = output_gradient[idx];
    }
}

__global__ void update_weights_kernel(float *weights, float *weight_gradients, float *biases, float *bias_gradients, float learning_rate, int input_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < output_size) {
        for (int j = 0; j < input_size; ++j) {
            weights[idx * input_size + j] -= learning_rate * weight_gradients[idx * input_size + j];
        }
        biases[idx] -= learning_rate * bias_gradients[idx];
    }
}

__global__ void cross_entropy_loss_kernel(float* output, float* target, float* loss, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Mỗi thread tính loss cho một phần tử
        float result = -target[idx] * log(output[idx]);  // Tính mất mát cho phần tử idx
        atomicAdd(loss, result);  // Cộng dồn mất mát
    }
}

__global__ void cross_entropy_loss_gradient_kernel(float* output, float* target, float* gradient, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Tính toán gradient của Cross-Entropy Loss
        gradient[idx] = output[idx] - target[idx];
    }
}
