#include "hip/hip_runtime.h"
#include "Kernel.h"
#include <cmath>
#include <float.h>

__global__ void forward_kernel(float *input, float *output, float *weights, float *biases, int input_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < output_size) {
        // Tính toán tổng có trọng số (linear transformation)
        float sum = 0.0f;
        for (int j = 0; j < input_size; ++j) {
            sum += weights[idx * input_size + j] * input[j];
        }
        sum += biases[idx];
        output[idx] = sum;  // Không áp dụng hàm kích hoạt ở đây
    }
}

__global__ void relu_kernel(float *input, float *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Áp dụng hàm kích hoạt ReLU
        output[idx] = (input[idx] > 0) ? input[idx] : 0;  // ReLU: f(x) = max(0, x)
    }
}

__global__ void softmax_kernel(float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        // Tìm giá trị max(x) để cải thiện độ ổn định số học
        float max_val = -FLT_MAX;
        for (int i = 0; i < size; ++i) {
            max_val = fmaxf(max_val, input[i]);
        }

        // Tính e^(input[i] - max_val)
        output[idx] = expf(input[idx] - max_val);
    }

    __syncthreads();  // Đồng bộ hóa các thread trong block

    // Tính tổng e^(input[i] - max_val)
    __shared__ float sum_exp;
    if (idx == 0) {
        sum_exp = 0.0f;
        for (int i = 0; i < size; ++i) {
            sum_exp += output[i];
        }
    }

    __syncthreads();  // Đồng bộ hóa sau khi tính tổng

    // Chuẩn hóa Softmax
    if (idx < size) {
        output[idx] /= sum_exp;
    }
}


__global__ void backward_kernel(float *input, float *output_gradient, float *weights, float *weight_gradients, float *bias_gradients, int input_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < output_size) {
        for (int j = 0; j < input_size; ++j) {
            weight_gradients[idx * input_size + j] = output_gradient[idx] * input[j];
        }
        bias_gradients[idx] = output_gradient[idx];
    }
}

__global__ void update_weights_kernel(float *weights, float *weight_gradients, float *biases, float *bias_gradients, float learning_rate, int input_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < output_size) {
        for (int j = 0; j < input_size; ++j) {
            weights[idx * input_size + j] -= learning_rate * weight_gradients[idx * input_size + j];
        }
        biases[idx] -= learning_rate * bias_gradients[idx];
    }
}

__global__ void cross_entropy_loss_kernel(float* output, float* target, float* loss, float* gradient, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Tính toán mất mát (loss) cho mỗi phần tử (cross-entropy)
        float result = -target[idx] * log(output[idx]);  // Tính mất mát cho phần tử idx
        atomicAdd(loss, result);  // Cộng dồn mất mát
        gradient[idx] = output[idx] - target[idx];  // Gradient cho Cross-Entropy loss (sau softmax)
    }
}

__global__ void cross_entropy_loss_gradient_kernel(float* output, float* target, float* gradient, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Tính toán gradient của Cross-Entropy Loss
        gradient[idx] = output[idx] - target[idx];
    }
}
