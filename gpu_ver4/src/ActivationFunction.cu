#include "hip/hip_runtime.h"
#include "ActivationFunction.h"
#include "Kernel.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include "Macro.h"
#include <hip/hip_fp16.h>  // Thêm thư viện để làm việc với half precision

void ReLU::activate(half* input, half* output, int size) const {
    half *d_input, *d_output;

    CHECK(hipMalloc(&d_input, size * sizeof(half)));
    CHECK(hipMalloc(&d_output, size * sizeof(half)));

    // Chuyển từ half sang float trước khi sao chép vào bộ nhớ của GPU
    float* temp_input = new float[size];
    for (int i = 0; i < size; ++i) {
        temp_input[i] = __half2float(input[i]);
    }
    CHECK(hipMemcpy(d_input, temp_input, size * sizeof(float), hipMemcpyHostToDevice));
    delete[] temp_input;

    int blocks = (size + 255) / 256;
    relu_kernel<<<blocks, 256>>>(d_input, d_output, size);
    CHECK(hipDeviceSynchronize());

    // Chuyển lại từ half sang float khi sao chép kết quả về host
    float* temp_output = new float[size];
    CHECK(hipMemcpy(temp_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < size; ++i) {
        output[i] = __float2half(temp_output[i]);
    }
    delete[] temp_output;

    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
}

void ReLU::derivative(half* output, half* d_output, int size) const {
    half *d_out, *d_dout;

    CHECK(hipMalloc(&d_out, size * sizeof(half)));
    CHECK(hipMalloc(&d_dout, size * sizeof(half)));

    // Chuyển từ half sang float trước khi sao chép vào bộ nhớ của GPU
    float* temp_out = new float[size];
    float* temp_dout = new float[size];
    for (int i = 0; i < size; ++i) {
        temp_out[i] = __half2float(output[i]);
        temp_dout[i] = __half2float(d_output[i]);
    }
    CHECK(hipMemcpy(d_out, temp_out, size * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_dout, temp_dout, size * sizeof(float), hipMemcpyHostToDevice));
    delete[] temp_out;
    delete[] temp_dout;

    int blocks = (size + 255) / 256;
    relu_derivative_kernel<<<blocks, 256>>>(d_out, d_dout, size);
    CHECK(hipDeviceSynchronize());

    // Chuyển lại từ half sang float khi sao chép kết quả về host
    CHECK(hipMemcpy(temp_dout, d_dout, size * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < size; ++i) {
        d_output[i] = __float2half(temp_dout[i]);
    }
    delete[] temp_dout;

    CHECK(hipFree(d_out));
    CHECK(hipFree(d_dout));
}

void Softmax::activate(half* input, half* output, int size) const {
    half *d_input, *d_output;

    CHECK(hipMalloc(&d_input, size * sizeof(half)));
    CHECK(hipMalloc(&d_output, size * sizeof(half)));

    // Chuyển từ half sang float trước khi sao chép vào bộ nhớ của GPU
    float* temp_input = new float[size];
    for (int i = 0; i < size; ++i) {
        temp_input[i] = __half2float(input[i]);
    }
    CHECK(hipMemcpy(d_input, temp_input, size * sizeof(float), hipMemcpyHostToDevice));
    delete[] temp_input;

    softmax_kernel<<<1, 1>>>(d_input, d_output, size);
    CHECK(hipDeviceSynchronize());

    // Chuyển lại từ half sang float khi sao chép kết quả về host
    float* temp_output = new float[size];
    CHECK(hipMemcpy(temp_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < size; ++i) {
        output[i] = __float2half(temp_output[i]);
    }
    delete[] temp_output;

    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
}

void Softmax::derivative(half* output, half* d_output, int size) const {
    // Softmax + CrossEntropy: derivative trực tiếp = output - target (đã xử lý bên ANN::backward)
    // Không làm gì ở đây
}
