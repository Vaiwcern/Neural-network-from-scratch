#include "hip/hip_runtime.h"
#include "Kernel.h"
#include <cmath>


__global__ void forward_kernel(float *input, float *output, float *weights, float *biases, 
                               int input_size, int output_size, int batch_size) {
    // Mỗi block xử lý một sample (b)
    int b = blockIdx.x;
    if (b >= batch_size) return;

    // Mỗi thread xử lý một neuron output (o)
    int o = threadIdx.x;
    if (o >= output_size) return;

    // Shared memory để lưu tile input
    extern __shared__ float s_input[];

    float sum = biases[o];

    // Chia input_size thành từng tile có kích thước = output_size (hoặc nhỏ hơn ở tile cuối)
    for (int i = 0; i < input_size; i += output_size) {
        int idx = i + o;
        float val = 0.0f;
        if (idx < input_size) {
            val = input[b * input_size + idx];
        }

        // Load giá trị input vào shared memory
        s_input[o] = val;
        __syncthreads();

        int tile_size = min(output_size, input_size - i);

        // Tính dot product trên tile vừa load
        // Mỗi thread xử lý neuron o, nên lấy weights tương ứng: weights[o * input_size + ...]
        for (int k = 0; k < tile_size; ++k) {
            sum += weights[o * input_size + (i + k)] * s_input[k];
        }
        __syncthreads();
    }

    // Ghi output
    output[b * output_size + o] = sum;
}


__global__ void backward_kernel(
    float *input, 
    float *output_gradient, 
    float *weights, 
    float *weight_gradients, 
    float *bias_gradients, 
    float *input_gradient,
    int input_size, 
    int output_size, 
    int batch_size
) {
    // Each block handles one output neuron
    int o = blockIdx.x;
    if (o >= output_size) return;

    // Each thread handles one input neuron
    int j = threadIdx.x;
    if (j >= input_size) return;

    // Shared memory for weights of the current output neuron
    extern __shared__ float s_weights[];

    // Load weights into shared memory
    if (j < input_size) {
        s_weights[j] = weights[o * input_size + j];
    }
    __syncthreads();

    float wgrad = 0.0f;

    // Compute weight gradients and accumulate bias gradients
    for (int b = 0; b < batch_size; b++) {
        float grad = output_gradient[b * output_size + o];
        float inp = input[b * input_size + j];
        wgrad += grad * inp;

        // Update input gradients with atomic operations
        float grad_input = s_weights[j] * grad;
        atomicAdd(&input_gradient[b * input_size + j], grad_input);
    }

    // Store the computed weight gradient
    weight_gradients[o * input_size + j] = wgrad;

    // Compute and store bias gradient (only once per output neuron)
    if (j == 0) {
        float total_bgrad = 0.0f;
        for (int b = 0; b < batch_size; b++) {
            total_bgrad += output_gradient[b * output_size + o];
        }
        atomicAdd(&bias_gradients[o], total_bgrad);
    }
}

__global__ void relu_kernel(float *input, float *output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        output[idx] = (input[idx] > 0) ? input[idx] : 0;
    }
}

__global__ void relu_derivative_kernel(float *output, float *d_output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float grad = (output[idx] > 0) ? 1.0f : 0.0f;
        d_output[idx] *= grad;
    }
}

__global__ void softmax_kernel(float *input, float *output, int size) {
    // Tính trên 1 vector
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        float max_val = -INFINITY;
        for (int i = 0; i < size; i++) {
            if (input[i] > max_val) max_val = input[i];
        }

        float sum_exp = 0.0f;
        for (int i = 0; i < size; i++) {
            float val = expf(input[i] - max_val);
            output[i] = val;
            sum_exp += val;
        }

        for (int i = 0; i < size; i++) {
            output[i] /= sum_exp;
        }
    }
}


__global__ void update_weights_kernel(float *weights, float *weight_gradients, float *biases, float *bias_gradients, float learning_rate, int input_size, int output_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Kiểm tra nếu thread đang xử lý idx hợp lệ
    if (idx < output_size) {
        // Sử dụng shared memory để lưu trữ gradient tạm thời trong mỗi block
        extern __shared__ float shared_weights[];

        // Đảm bảo rằng không vượt quá kích thước shared memory
        // Tối ưu cho kích thước nhỏ hơn hoặc bằng 256
        float* shared_gradients = shared_weights + input_size; 

        // Tải dữ liệu vào shared memory, mỗi thread sẽ xử lý một phần trọng số và gradient
        for (int j = threadIdx.x; j < input_size; j += blockDim.x) {
            shared_weights[j] = weights[idx * input_size + j];  // Cập nhật trọng số trong shared memory
            shared_gradients[j] = weight_gradients[idx * input_size + j];  // Cập nhật gradient trong shared memory
        }

        // Đồng bộ hóa threads trong block để đảm bảo tất cả dữ liệu đã được tải vào shared memory
        __syncthreads();

        // Cập nhật trọng số và gradient trong shared memory
        for (int j = 0; j < input_size; ++j) {
            shared_weights[j] -= learning_rate * shared_gradients[j];
        }

        // Cập nhật lại trọng số trong global memory
        for (int j = threadIdx.x; j < input_size; j += blockDim.x) {
            weights[idx * input_size + j] = shared_weights[j];  // Lưu lại trọng số đã cập nhật vào global memory
        }

        // Cập nhật bias trong global memory
        if (threadIdx.x == 0) {  // Chỉ một thread trong block thực hiện cập nhật bias
            biases[idx] -= learning_rate * bias_gradients[idx];
        }
    }
}
