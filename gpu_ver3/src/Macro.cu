#include "Macro.h"

// Định nghĩa các hàm thành viên của GpuTimer
GpuTimer::GpuTimer()
{
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
}

GpuTimer::~GpuTimer()
{
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
}

void GpuTimer::Start()
{
    CHECK(hipEventRecord(start, 0));
    CHECK(hipEventSynchronize(start));
}

void GpuTimer::Stop()
{
    CHECK(hipEventRecord(stop, 0));
}

float GpuTimer::Elapsed()
{
    float elapsed;
    CHECK(hipEventSynchronize(stop));
    CHECK(hipEventElapsedTime(&elapsed, start, stop));
    return elapsed;
}
