#include "ActivationFunction.h"
#include "Kernel.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <iostream>
#include "Macro.h"
#include <hip/hip_fp16.h>  // Thư viện hỗ trợ kiểu dữ liệu half

void ReLU::activate(half* input, half* output, int size) const {
    half *d_input, *d_output;

    CHECK(hipMalloc(&d_input, size * sizeof(half)));
    CHECK(hipMalloc(&d_output, size * sizeof(half)));

    CHECK(hipMemcpy(d_input, input, size * sizeof(half), hipMemcpyHostToDevice));

    int blocks = (size + 255) / 256;
    relu_kernel<<<blocks, 256>>>(d_input, d_output, size);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(output, d_output, size * sizeof(half), hipMemcpyDeviceToHost));
    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
}

void ReLU::derivative(half* output, half* d_output, int size) const {
    half *d_out, *d_dout;

    CHECK(hipMalloc(&d_out, size * sizeof(half)));
    CHECK(hipMalloc(&d_dout, size * sizeof(half)));

    CHECK(hipMemcpy(d_out, output, size * sizeof(half), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_dout, d_output, size * sizeof(half), hipMemcpyHostToDevice));

    int blocks = (size + 255) / 256;
    relu_derivative_kernel<<<blocks, 256>>>(d_out, d_dout, size);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(d_output, d_dout, size * sizeof(half), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_out));
    CHECK(hipFree(d_dout));
}

void Softmax::activate(half* input, half* output, int size) const {
    half *d_input, *d_output;

    CHECK(hipMalloc(&d_input, size * sizeof(half)));
    CHECK(hipMalloc(&d_output, size * sizeof(half)));

    CHECK(hipMemcpy(d_input, input, size * sizeof(half), hipMemcpyHostToDevice));

    softmax_kernel<<<1, 1>>>(d_input, d_output, size);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(output, d_output, size * sizeof(half), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_input));
    CHECK(hipFree(d_output));
}

void Softmax::derivative(half* output, half* d_output, int size) const {
    // Softmax + CrossEntropy: derivative trực tiếp = output - target (đã xử lý bên ANN::backward)
    // Không làm gì ở đây
}
